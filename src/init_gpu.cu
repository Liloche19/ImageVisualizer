#include "../include/visualizer_cuda.cuh"
#include <pthread.h>

void *load_drivers(void *data)
{
    void *ptr = NULL;

    hipMalloc(&ptr, 1);
    hipDeviceSynchronize();
    hipFree(ptr);
    return NULL;
}

extern "C" void init_gpu(pthread_t *gpu_loader)
{
    int device = 0;
    hipError_t err = hipGetDeviceCount(&device);

    if (err != hipSuccess || device == 0)
        return;
    pthread_create(gpu_loader, NULL, &load_drivers, NULL);
    return;
}
