#include "../include/visualizer_cuda.cuh"
#include <pthread.h>

void *load_drivers(void *data)
{
    void *ptr = NULL;

    hipMalloc(&ptr, 1);
    hipDeviceSynchronize();
    hipFree(ptr);
    return NULL;
}

extern "C" void init_gpu(pthread_t *gpu_loader)
{
    int device = 0;
    hipError_t err = hipGetDeviceCount(&device);

    if (err != hipSuccess || device == 0)
        return;
    if (pthread_create(gpu_loader, NULL, &load_drivers, NULL) != 0) {
        fprintf(stderr, "Error while creating thread!\n");
        exit(1);
    }
    return;
}
