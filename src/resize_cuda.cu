#include "hip/hip_runtime.h"
#include "../include/visualizer_cuda.cuh"

extern __global__ void resize_image_cuda(Screen *screen, Image *image, float ratio_x, float ratio_y);

extern "C" int resize_cuda(Screen *screen, Image *image, float ratio_x, float ratio_y)
{
    int device = 0;
    hipError_t err = hipGetDeviceCount(&device);
    int nb_pixels = 0;
    int nb_blocks = 0;
    int image_size = 0;
    Screen *gpu_screen;
    Image *gpu_image;

    if (err != hipSuccess || device == 0)
        return 1;
    nb_pixels = screen->cols * screen->rows;
    nb_blocks = (nb_pixels + CUDA_BLOCK_SIZE - 1) / CUDA_BLOCK_SIZE;
    screen->buffer_size = sizeof(char) * (sizeof(PIXEL_TEMPLATE) * screen->cols * screen->rows + (sizeof(RESET) + 1) * screen->rows);
    image_size = sizeof(unsigned char) * image->channels * image->height * image->width;
    if (pthread_join(screen->gpu_loader, NULL) != 0) {
        fprintf(stderr, "Error while waiting thread!\n");
        exit(1);
    }
    if (hipMalloc(&gpu_screen, sizeof(Screen)) != hipSuccess || hipMalloc(&gpu_image, sizeof(Image)) != hipSuccess) {
        fprintf(stderr, "Error initialising structures!\n");
        exit(1);
    }
    if (hipMalloc(&(screen->gpu_print_buffer), screen->buffer_size) != hipSuccess || hipMalloc(&(image->gpu_pixels), image_size) != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        exit(1);
    }
    if (hipMemcpy(image->gpu_pixels, image->pixels, image_size, hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "hipMemcpy to device failed!\n");
        exit(1);
    }
    if (hipMemcpy(gpu_screen, screen, sizeof(Screen), hipMemcpyHostToDevice) != hipSuccess || hipMemcpy(gpu_image, image, sizeof(Image), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Error copying data to GPU!\n");
        exit(1);
    }
    resize_image_cuda<<<nb_blocks, CUDA_BLOCK_SIZE>>>(gpu_screen, gpu_image, ratio_x, ratio_y);
    if ((err = hipDeviceSynchronize()) != hipSuccess) {
        fprintf(stderr, "sync failed!\n%s\n", hipGetErrorString(err));
        exit(1);
    }
    if ((err = hipMemcpy(screen->print_buffer, screen->gpu_print_buffer, screen->buffer_size, hipMemcpyDeviceToHost)) != hipSuccess) {
        fprintf(stderr, "hipMemcpy to host failed!\n%s\n", hipGetErrorString(err));
        exit(1);
    }
    hipFree(screen->gpu_print_buffer);
    hipFree(image->gpu_pixels);
    hipFree(gpu_screen);
    hipFree(gpu_image);
    return 0;
}
